#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#define TUP 25
#define TDOWN 250
#define TLEFT 25
#define TRIGHT 25

#define ITERATIONS 250


int length = 2;
int height = 4;
float spacing = 0.1;
int size_x = (length/spacing) + 1;
int size_y = (height/spacing);

// TIMER %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
double get_walltime() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return (double) (tp.tv_sec + tp.tv_usec*1e-6);
}

void initTemps(float **T) {
    for(int i=0; i<size_x; i++) {
        T[i][0] = TDOWN;
        T[i][size_y-1] = TUP;
    }
    for(int j=0; j<size_y; j++) {
        T[0][j] = TLEFT;
        T[size_x-1][j] = TRIGHT;
    }
}

__global__ void temp(float *T,float *TNew)
{
        int j = threadIdx.x;
        int i = blockIdx.x;
        TNew[i][j] = 0.25 * (T[i+1][j] + T[i-1][j] + T[i][j+1] + T[i][j-1]);
}


__global__ void temp1(float *T, float *TNew)
{
        int j = threadIdx.x;
        int i = blockIdx.x;
        T[i][j] = TNew[i][j];
}


int main() {
    printf("Enter width of the plate: ");
    scanf("%d", &length);
    printf("Enter the height of the plate: ");
    scanf("%d", &height);
    printf("Enter spacing between nodes: ");
    scanf("%f", &spacing);
    size_x = (length/spacing) + 1;
    size_y = (height/spacing);

    float **T;
    hipMalloc((void**)&T, size_x*size_y*sizeof(float));

    float **TNew;
    hipMalloc((void**)&TNew, size_x*size_y*sizeof(float));


    for(int i=0; i<size_x*size_y; i++) {
            T[i][j] = 0;
            TNew[i][j] = 0;
    }
    initTemps(T);
    initTemps(TNew);

    float omega = 1.5, error = 1.0;

    double time1 = get_walltime(), time2;

    for(int k=0; k<ITERATIONS; k++) {

        temp<<size_x,size_y>>(T, TNew);

        temp1<<size_x,size_y>>(T, TNew);
  }
    time2 = get_walltime();
    printf("\nTime: %fs\n", time2-time1);


    //    printf("\nResults:\n");
    //    for(int j=size_y-1; j>=0; j--) {
    //        for(int i=0; i<size_x; i++) {
    //            printf("%f\t",T[i][j]);
    //        }
    //        printf("\n");
    //    }

    FILE *fp1 = fopen("resultCoordinatesAllTemp.txt", "w");
    FILE *fp2 = fopen("resultMatrixAllTemp.txt", "w");

    //    printf("\nTEMPERATURE COORDINATES:\n");
    float xcoord = 0.0;
    float ycoord = 0.0;
    for(int j=0; j<size_y; j++, ycoord += spacing){
        for(int i=0; i<size_x; i++, xcoord += spacing) {
            fprintf(fp1, "%f\t%f\t%f\n", xcoord, ycoord, T[i][j]);
        }
        xcoord = 0.0;
    }
    //    printf("\nTEMPERATURES:\n");
    for(int j=size_y-1; j>=0; j--) {
        for(int i=0; i<size_x; i++) {
            fprintf(fp2, "%f\t", T[i][j]);
        }
        fprintf(fp2, "\n");
    }

//    for(int k=0; k<size_x; k++) {
//        free(T[k]);
//        free(TNew[k]);
//    }
//    free(T);
//    free(TNew);
    fclose(fp1);
    fclose(fp2);
    return 0;
}
                                                                
                                                                                                                                                                                          86,1-8        47%
